#include "hip/hip_runtime.h"
//
//  gpu_lrt.cu
//  LRT
//
//  Copyright (c) 2013 The University of Sydney. All rights reserved.
//
//  GPU implementation of LRT in CUDA
//


#include <hip/hip_runtime.h> 
#include <assert.h> 
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string>
#include <sstream>
#include <stdexcept>
#include <iostream> 

extern "C" {
#include "lrt.h"
#include "prefix.h"
#include "gpu.h"
#include "heap.h"
}

//
// compare two rectangles. If the scores are equal, the coordinates
// are used to create a total order 
// 
__device__ __forceinline__ int rectangle_compare(rectangle *r_a, rectangle *r_b)
{

   if(isnan(r_a->score)) return -1;
   if(isnan(r_b->score)) return 1;
   
   float score_a = r_a->score;
   float score_b = r_b->score;

   if (score_a < score_b) return -1;
   if (score_a > score_b) return 1;
   
   int   area_a=(r_a->j2-r_a->j1+1)*(r_a->i2-r_a->i1+1);
   int   area_b=(r_b->j2-r_b->j1+1)*(r_b->i2-r_b->i1+1);

   if(area_a < area_b) return 1;
   if(area_a > area_b) return -1;
 
   if(r_a->i1 < r_b->i1) return 1;
   if(r_a->i1 > r_b->i1) return -1;

   if(r_a->i2 < r_b->i2) return 1;
   if(r_a->i2 > r_b->i2) return -1;

   if(r_a->j1 < r_b->j1) return 1;
   if(r_a->j1 > r_b->j1) return -1;

   if(r_a->j2 < r_b->j2) return 1;
   if(r_a->j2 > r_b->j2) return -1;

   return 0;
}

//
// raises the last element of the min-heap until heap condition holds
//
__device__  __forceinline__ void rheaprise(rectangle *heap, int n)
{
   // allocate memory for swap on heap
   rectangle tmp;
   int k;

   for (k=n-1;k > 0; ) { 
       // set j to parent
       int j = k/2;
       // check whether a[k] is smaller than parent a[j] 
       if (rectangle_compare(&heap[k],&heap[j]) < 0) { 
          // swap elements k and j 
          tmp=heap[k];
          heap[k]=heap[j];
          heap[j]=tmp;

          // set k to current node 
          k = j; 
       } else  break;
   }   
}

//
// sinks the first element of the min-heap until heap condition holds
//
__device__  __forceinline__ void rheapsink(rectangle *heap, int n)
{
   // allocate memory for swap on heap
   rectangle tmp;
   int k;
   for (k=0;2*k<n;) {

      // set j to first child of k
      int j = 2 * k + 1;

      // check which of the children (if there are two) is smaller
      // move j to the smaller child so that the smaller child is 
      // swapped with a[k]
      if (j < n-1) {
         if (rectangle_compare(&heap [j+1], &heap[j]) < 0) {
            j++;
         }
      }
      // check whether min-mheap condition holds, i.e a[k] is smaller 
      // than smallest child. If so, stop.
      if (rectangle_compare(&heap[k], &heap[j]) <0)
         break;
      // swap elements k and j 
       tmp=heap[k];
       heap[k]=heap[j];
       heap[j]=tmp;
       // set new n
       k = j;
   }
}


//
// execute LRT (fully enumeration mapping+heap) on GPGPU
//
__global__ void lrt_kernel(int width, 
                           int height,
                           int kbest, 
                           cell *pA, 
                           cell *pB, 
                           cell *pX,  
                           cell *pY, 
                           rectangle *r_global, 
                           int *heapsize_global
                          )
{
   // compute grid totals, ratio, and likelihood
   int g_n = pA[width * height - 1].n;
   int g_k = pA[width * height - 1].k;
   float g_q = (float)g_k / (float)g_n;
   float g_l = g_k * log(g_q);

   // rectangular length
   int rx_size = (width + 1)/2;
   int ry_size = (height + 1)/2;

   // compute the number of iterations necessary for the loops of a thread
   // (for each dimension)
   int tx_size=((width+1)*rx_size + blockDim.x*gridDim.x-1)/(blockDim.x*gridDim.x);
   int ty_size=((height+1)*ry_size + blockDim.y*gridDim.y-1)/(blockDim.y*gridDim.y);
 
   // allocate heap with size k for each thread  locally
   extern __shared__ rectangle sr[];
   rectangle *r = &sr[(threadIdx.y * blockDim.x + threadIdx.x)*kbest]; 
   
   // heap size (initialized with zero and grows up to kbest)
   int heap_size=0;  

   // thread loops 
   int x,y;
   for (x=0;x<tx_size;x++) { 
      for (y=0;y<ty_size;y++) { 
      
         // convert grid and block coordinates back to 
         // a four dimensional grid.
         int grid_x=(blockIdx.x*blockDim.x+threadIdx.x)*tx_size+x;
         int grid_y=(blockIdx.y*blockDim.y+threadIdx.y)*ty_size+y;

         // check whether thread is out of range 
         // (if block sizes don't divide grid length) 
         if(grid_x>=(width+1)*rx_size) continue;
         if(grid_y>=(height+1)*ry_size) continue;
  
         // get the four dimensional rectangular 
         // coordinates using inverse Horner scheme. 
         int ti1=(grid_y)%ry_size;
         int ti2=(grid_y)/ry_size;
         int tj1=(grid_x)%rx_size;
         int tj2=(grid_x)/rx_size;

         // transform rectangular coordinates to 
         // triangular coordinates. 
         int i1,j1,i2,j2;
         if ((tj2 < width - tj1)) {
            j1=tj1;
            j2=tj2+tj1;
         } else if( (tj1+1)*2 < width+1){
            j1=width-tj1-1;
            j2=width-tj2+j1;
         } else continue;
         if ((ti2 < height - ti1)) {
             i1=ti1;
             i2=ti2+ti1;
         } else if( (ti1+1)*2 < height+1){
             i1=height-ti1-1;
             i2=height-ti2+i1;
         } else continue;

         // compute rectangle totals, ratio and likelihood
         int   a_n = pA[I(i2,j2)].n;
         int   a_k = pA[I(i2,j2)].k;
         int   b_n = pB[I(i1,j1)].n;
         int   b_k = pB[I(i1,j1)].k;
         int   y_n = pY[I(i1,j2)].n;
         int   y_k = pY[I(i1,j2)].k;
         int   x_n = pX[I(i2,j1)].n;
         int   x_k = pX[I(i2,j1)].k;
         int   r_n = a_n + b_n + x_n + y_n - g_n;
         int   r_k = a_k + b_k + x_k + y_k - g_k;
         float r_q = (float)r_k / (float)r_n;
         float r_l = r_k * log(r_q) - r_k; 
  
         // compute rectangle's complement totals, ratio, and likelihood
         int   c_n = g_n - r_n;
         int   c_k = g_k - r_k;
         float c_q = (float)c_k / (float)c_n;
         float c_l = c_k * log(c_q) - c_k;
  
         // compute score 
         float score = r_l + c_l - g_l; 

         // populate current rectangle
         rectangle current;
         current.score = score; 
         current.i1 = i1;
         current.j1 = j1;
         current.i2 = i2;
         current.j2 = j2;
     
         // store result in heap
         // if the heap size is still smaller than kbest, add rectangle to the end of the heap
         // and rise the last element until the heap condition holds 
         if (heap_size < kbest) { 
            r[heap_size].score = score; 
            r[heap_size].i1 = i1;
            r[heap_size].j1 = j1;
            r[heap_size].i2 = i2;
            r[heap_size].j2 = j2;
            heap_size = heap_size + 1;
            rheaprise(r,heap_size);
         // otherwise check whether the current score is greater than the smallest element (=first element) 
         // if so, replace the first element with current rectangle and sink the current element until 
         // the heap condition holds. 
         } else if (rectangle_compare(&current,r) > 0) {
            r[0] = current;
            rheapsink(r,kbest);
         } 
      }
   }

   // store heap_size
   heapsize_global[((blockIdx.y*blockDim.y+threadIdx.y)*gridDim.x*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x)] = heap_size;

   // Write heap back to global memory
   memcpy(&r_global[((blockIdx.y*blockDim.y+threadIdx.y)*gridDim.x*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x)*kbest],r,heap_size*sizeof(rectangle));
}


//
// gpu simulation of LRT computation
// 
__host__ rectangle *gpu_lrt(grid *g,int kbest)
{

   // set width and height
   int width = g->width;
   int height = g->height;
   
   // compute prefix sums
   prefix_sums *p = compute_psums(g);

   // allocate memory for prefix sums 
   // and transfer them to the CUDA device
   cell *pa_d,
        *pb_d, 
        *px_d, 
        *py_d;
   size_t num_cells=sizeof(cell)*width*height;
   hipMalloc(&pa_d,num_cells);
   hipMalloc(&pb_d,num_cells);
   hipMalloc(&px_d,num_cells);
   hipMalloc(&py_d,num_cells);
   hipMemcpy(pa_d,p->A,num_cells,hipMemcpyHostToDevice);
   hipMemcpy(pb_d,p->B,num_cells,hipMemcpyHostToDevice);
   hipMemcpy(px_d,p->X,num_cells,hipMemcpyHostToDevice);
   hipMemcpy(py_d,p->Y,num_cells,hipMemcpyHostToDevice);
  
   // allocate memory for resulting heaps for each thread 
   // and heap sizes.  
   rectangle *r_global_d; 
   hipMalloc(&r_global_d,sizeof(rectangle)*kbest*THREADS_IN_GRID);
   int *heapsize_global_d; 
   hipMalloc(&heapsize_global_d,sizeof(int)*THREADS_IN_GRID);

   // run kernel
   dim3 dimg(GRIDSIZE_X,GRIDSIZE_Y);
   dim3 dimb(BLOCKSIZE_X,BLOCKSIZE_Y);
   lrt_kernel<<<dimg,dimb,sizeof(rectangle)*BLOCKSIZE_X*BLOCKSIZE_Y*kbest>>>(
      width,
      height,
      kbest,
      pa_d, 
      pb_d,
      px_d,
      py_d,  
      r_global_d,
      heapsize_global_d);

   // retrieve result from GPGPU
   rectangle *r_global = ALLOCV(rectangle, sizeof(rectangle)*kbest*THREADS_IN_GRID);
   int *heapsize_global = ALLOCV(int, sizeof(int) * THREADS_IN_GRID); 
   hipMemcpy(r_global,r_global_d,sizeof(rectangle)*kbest*THREADS_IN_GRID,hipMemcpyHostToDevice);
   hipMemcpy(heapsize_global,heapsize_global_d,sizeof(int)*THREADS_IN_GRID,hipMemcpyHostToDevice);

   ////////////////////////////////////////////////////////////////////////////////////
   // join heaps of all threads to a single heap 
   rectangle *r = ALLOCV(rectangle,kbest);
   int heap_size = 0;

   // enumerate all blocks 
   int i;
   for (i=0;i<THREADS_IN_GRID;i++) { 
      int hs = heapsize_global[i];
      rectangle *rs = &r_global[i*kbest];
      int j;
      for (j=0;j<hs;j++) {
         if (heap_size < kbest) { 
            r[heap_size] = rs[j];
            heap_size = heap_size + 1;
            minheap_rise(r,heap_size,sizeof(rectangle),heap_rcmp);
         } else if (heap_rcmp(&rs[j],r) > 0) {
            r[0] = rs[j];
            minheap_sink(r,kbest,sizeof(rectangle),heap_rcmp);
         } 
      } 
   } 

   // sort result and return it
   qsort(r,heap_size,sizeof(rectangle),rectangle_compare);  

   return r; 
}
