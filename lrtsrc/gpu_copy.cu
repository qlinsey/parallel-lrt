#include "hip/hip_runtime.h"
//
//  real gpu_lrt.cu
//  LRT
//
//  Copyright (c) 2013 The University of Sydney. All rights reserved.
//
//  gpu lrt computation with range mapping scheme and heap reduction 
//


#include <hip/hip_runtime.h> 
#include <assert.h> 
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
//#include "util/cuPrintf.cu" 
#include <string>
#include <sstream>
#include <stdexcept>
#include <iostream> 


using namespace std;

#ifndef __Kernel_CU__
#define __Kernel_CU__
__global__void kernel(int a ){
  int x=threadIdx.x+a;
  int y=threadIdx.y+a;

}
extern "C" void wrapper_lrt(){

   kernel<<<2,2>>>(2);

}
#endif
////////////////////////////////////////////////////////////////////////////
//  device functions
//

//
// sinks the first element of the min-heap until heap condition holds
//

/*__device__ void minheap_sink(int t){


}*/

//
// fast GPGPU implementation of LRT using prefix sums 
//
/*__global__ void gpu_lrt(prefix_sums *p, rectangle * r_global, int * heapsize_global,  int tx_size, int ty_size, int width, int height, int kbest, int g_n, int g_k, float g_q, float g_l) {
   // rectangular length
   int rx_size = (width + 1)/2;
   int ry_size = (height + 1)/2;
   int heap_size=0;
   int x,y;
   // allocate heap with size k for each thread  locally
   // in cuda replace with "rectangle r[kbest];"
   //rectangle* r =(rectangle *)malloc(sizeof(rectangle)*kbest);
   rectangle r[10];
   for(x=0;x<tx_size;x++) { 
      for (y=0;y<ty_size;y++) { 
         // convert grid and block coordinates back to 
         // a four dimensional grid.
         int grid_x=blockIdx.x*blockDim.x*tx_size+threadIdx.x*tx_size+x;
         int grid_y=blockIdx.y*blockDim.y*ty_size+threadIdx.y*ty_size+y;
         // check whether thread is out of range 
         // (if block sizes don't divide grid length) 
         if(grid_x>=(width+1)*rx_size) continue;
         if(grid_y>=(height+1)*ry_size) continue;
         // get the four dimensional rectangular 
         // coordinates using inverse Horner scheme. 
         int ti1=(grid_y)%ry_size;
         int ti2=(grid_y)/ry_size;
         int tj1=(grid_x)%rx_size;
         int tj2=(grid_x)/rx_size;
//         assert(0 <= ti1 && ti1 < ry_size && 0 <= ti2 && ti2 < height+1 && "Rectangular coordinate for rows out of range");
//         assert(0 <= tj1 && tj1 < rx_size && 0 <= tj2 && tj2 < width+1 && "Rectangular coordinate for columns out of range");

                     // transform rectangular coordinates to 
                     // triangular coordinates. 
                     int i1,j1,i2,j2;
                     if ((tj2 < width - tj1)) {
                        j1=tj1;
                        j2=tj2+tj1;
                     } else if( (tj1+1)*2 < width+1){
                        j1=width-tj1-1;
                        j2=width-tj2+j1;
                     } else continue;
                     if ((ti2 < height - ti1)) {
                         i1=ti1;
                         i2=ti2+ti1;
                     } else if( (ti1+1)*2 < height+1){
                         i1=height-ti1-1;
                         i2=height-ti2+i1;
                     } else continue;

//                     assert(0 <= i1 && i1 <= i2 && i2 < height && "Triangular coordinate for rows out of range");
//                     assert(0 <= j1 && j1 <= j2 && j2 < width && "Triangular coordinate for columns out of range");
 
                     // compute rectangle totals, ratio and likelihood
                     int   a_n = p->A[I(i2,j2)].n;
                     int   a_k = p->A[I(i2,j2)].k;
                     int   b_n = p->B[I(i1,j1)].n;
                     int   b_k = p->B[I(i1,j1)].k;
                     int   y_n = p->Y[I(i1,j2)].n;
                     int   y_k = p->Y[I(i1,j2)].k;
                     int   x_n = p->X[I(i2,j1)].n;
                     int   x_k = p->X[I(i2,j1)].k;
                     int   r_n = a_n + b_n + x_n + y_n - g_n;
                     int   r_k = a_k + b_k + x_k + y_k - g_k;
                     float r_q = (float)r_k / (float)r_n;
                     float r_l = r_k * log(r_q) - r_k; 
              
                     // compute rectangle's complement totals, ratio, and likelihood
                     int   c_n = g_n - r_n;
                     int   c_k = g_k - r_k;
                     float c_q = (float)c_k / (float)c_n;
                     float c_l = c_k * log(c_q) - c_k;
              
                     // compute score 
                     float score = r_l + c_l - g_l; 

                     // populate current rectangle
                     rectangle current;
                     current.score = score; 
                     current.i1 = i1;
                     current.j1 = j1;
                     current.i2 = i2;
                     current.j2 = j2;
                 
                     // store result in heap
                     // if the heap size is still smaller than kbest, add rectangle to the end of the heap
                     // and rise the last element until the heap condition holds 
                     if (heap_size < kbest) { 
                        r[heap_size].score = score; 
                        r[heap_size].i1 = i1;
                        r[heap_size].j1 = j1;
                        r[heap_size].i2 = i2;
                        r[heap_size].j2 = j2;
                        heap_size = heap_size + 1;
                        //minheap_rise(r,heap_size,sizeof(rectangle),heap_rcmp);
                     // otherwise check whether the current score is greater than the smallest element (=first element) 
                     // if so, replace the first element with current rectangle and sink the current element until 
                     // the heap condition holds. 
                     } /*else if (heap_rcmp(&current,r) > 0) {
                        r[0] = current;
                        //minheap_sink(r,kbest,sizeof(rectangle),heap_rcmp);
                     } */
                  }
               }

               // store heap_size
               //blockIdx.y*blockDim.y*gridDim.x*blockDim.x+blockIdx.x*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x
               heapsize_global[((blockIdx.y*blockDim.y+threadIdx.y)*gridDim.x*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x)] = heap_size;
               // Write heap back to global memory
               memcpy(&r_global[((blockIdx.y*blockDim.y+threadIdx.y)*gridDim.x*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x)*kbest],r,heap_size*sizeof(rectangle));
}*/


// test gpu lrt code
/*int main(int argc, char* argv[]){
   // number of the best scoring rectangles printed by the output
   int kbest=DEFAULT_KBEST; 
   size_t size;         // number of rectangles in the grid
   //generate grid 
   grid      *g;           // input grid
   FILE *input = stdin;
    if (optind < argc){
        if (argc-optind > 1){
            error("only one input file permitted."); 
        }
        if ((input=fopen(argv[optind],"rb"))==NULL){
            error("cannot open input file %s.",optarg);
        }
    }
   g = read_grid(input);
   int width=g->width;
   int height=g->height;
   // compute prefix sums
   prefix_sums *p = compute_psums(g);
   // compute grid totals, ratio, and likelihood
   int g_n = p->A[width * height - 1].n;
   int g_k = p->A[width * height - 1].k;
   float g_q = (float)g_k / (float)g_n;
   float g_l = g_k * log(g_q);

   //block size ,means the number of theards in each block 
   int bx_size = 2;
   int by_size = 2;
   // define tile size for a thread
   int tx_size = 4; 
   int ty_size = 4; 
   // rectangular length
   int rx_size = (width + 1)/2;
   int ry_size = (height + 1)/2;
   int gx_size=((width+1)*rx_size + bx_size*tx_size-1)/(bx_size*tx_size);
   int gy_size=((height+1)*ry_size + by_size*ty_size-1)/(by_size*ty_size);
   
   // allocate a heap for each thread with size k 
   rectangle *r_global = (rectangle *)ALLOCV(rectangle,kbest*bx_size*by_size*gx_size*gy_size);
   int *heapsize_global = (int *)ALLOCV(int,bx_size*by_size*gx_size*gy_size);

   // A [(N+1],m]^2 grid needs to be mappeds to a two-dimensional
   // grid [(N+1)*m]^2 and divided by the block size. Ensure that if the
   // block size does not divide grid length, one extra block is added 
   // (NB: This is done by adding bx_size-1 before division). 

    size = (size_t)(g->width + 1) *
           (size_t)g->width  *
           (size_t)(g->height + 1) *
           (size_t)g->height / 4;
    // adjust kbest
    kbest = min(size, kbest);
    
   //dimg encodes the dimension of grid in terms of the number of
   //blocks in each dimension
   dim3 dimg(gx_size, gy_size);
   //dimb encodes the dimension of each block in terms of the number of threads per block 
   dim3 dimb(bx_size, by_size);
   gpu_lrt<<<dimg, dimb>>>(p, r_global,heapsize_global,tx_size,ty_size,width, height, kbest,g_n,g_k,g_q,g_l);
   
}*/
